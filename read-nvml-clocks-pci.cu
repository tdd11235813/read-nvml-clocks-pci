#include "hip/hip_runtime.h"
#include <nvml.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <string>
#include <stdexcept>
#include <iostream>
#include <algorithm>

#define CHECK_NVML(ans) check_nvml((ans), #ans, __FILE__, __LINE__)
#define CHECK_CUDA(ans) check_cuda((ans), #ans, __FILE__, __LINE__)

inline
void throw_error(int code,
                 const char* error_string,
                 const char* func,
                 const char* file,
                 int line) {
  std::cerr << error_string
            << " [" << code << "] "
            << file << ":" << line
            << " " << func
            << "\n";
  throw std::runtime_error("Error occurred.");
}

void check_cuda(hipError_t code,  const char *func, const char *file, int line) {
  if (code != hipSuccess) {
    throw_error(static_cast<int>(code),
                hipGetErrorString(code), func, file, line);
  }
}

void check_nvml(nvmlReturn_t code,  const char *func, const char *file, int line) {
  if (code != NVML_SUCCESS) {
    throw_error(static_cast<int>(code),
                nvmlErrorString(code), func, file, line);
  }
}

// [not used] simple cuda info
void cuda_info() {
  hipDeviceProp_t prop;
  size_t f=0, t=0;
  int ndevs = 0;
  CHECK_CUDA( hipGetDeviceCount(&ndevs) );
  for(int i=0; i<ndevs; ++i) {
    CHECK_CUDA( hipGetDeviceProperties(&prop, i) );
    CHECK_CUDA( hipMemGetInfo(&f, &t) );
    std::cout << i << ": " << prop.name << '"'
              << ", CC, " << prop.major << '.' << prop.minor
              << ", PCI Bus ID, " << prop.pciBusID
              << ", PCI Device ID, " << prop.pciDeviceID
              << ", Multiprocessors, "<< prop.multiProcessorCount
              << ", Memory [MiB], "<< t/1048576
              << ", MemoryFree [MiB], " << f/1048576
              << ", ECC enabled, " << prop.ECCEnabled
              << ", MemClock [MHz], " << prop.memoryClockRate/1000
              << ", GPUClock [MHz], " << prop.clockRate/1000
              << "\n"
      ;
  }
}

// simple nvml info
void nvml_info(std::vector<std::string>& bus_ids) {
  unsigned int device_count, i;

  CHECK_NVML( nvmlInit() );

  CHECK_NVML( nvmlDeviceGetCount(&device_count) );
  if(device_count<1)
    throw std::runtime_error("No NVML devices found.");
  bus_ids.resize(device_count);
  for (i = 0; i < device_count; i++)
  {
    char name[64];
    nvmlDevice_t device = 0;
    nvmlPciInfo_t pci;
//    unsigned int power = 0;
    CHECK_NVML( nvmlDeviceGetHandleByIndex(i, &device));
    CHECK_NVML( nvmlDeviceGetName(device, name, sizeof(name)/sizeof(name[0])));
//    CHECK_NVML( nvmlDeviceGetPowerUsage(device, &power)); // probably not supported on geforce,quadro
    CHECK_NVML( nvmlDeviceGetPciInfo(device, &pci));
    std::cout << i << ": Dev Name, " << name
              << ", PCI Bus (domain:bus:dev), " << pci.busId
              << ", PCI Bus, " << pci.bus
              << ", PCI Domain, " << pci.domain
              << ", PCI Dev, " << pci.device
              << ", PCI DevID, " << pci.pciDeviceId
              << ", PCI SubsysID, " << pci.pciSubSystemId
//              << ", Power (W), " << 0.001*power
              << "\n";
    bus_ids[i] = std::string(pci.busId);
  }

  CHECK_NVML(nvmlShutdown());
}


// [not used] demonstrate a coordinated cuda/nvml access
void test_cuda_and_nvml_device(int dev) {
  nvmlDevice_t device;
  hipDeviceProp_t prop;
  char name[NVML_DEVICE_NAME_BUFFER_SIZE];
  char pci_busid[NVML_DEVICE_PCI_BUS_ID_BUFFER_SIZE];
  CHECK_CUDA( hipGetDeviceProperties(&prop, dev) );

  CHECK_CUDA( hipDeviceGetPCIBusId( pci_busid, NVML_DEVICE_PCI_BUS_ID_BUFFER_SIZE, dev ) );
  CHECK_NVML( nvmlInit() );
  CHECK_NVML( nvmlDeviceGetHandleByPciBusId( pci_busid, &device ) );
  CHECK_NVML( nvmlDeviceGetName(device, name, NVML_DEVICE_NAME_BUFFER_SIZE) );
  bool equal = (strcmp(prop.name, name) == 0);
  std::cout << "Name (CUDA|NVML): " << prop.name << " | "<< name << " (" << (equal?"OK":"Mismatch") << ")\n";
  CHECK_NVML(nvmlShutdown());
}

void show_freqs(int dev, const std::vector<std::string> bus_ids) {
  nvmlDevice_t device = 0;
  hipDeviceProp_t prop;
  CHECK_CUDA( hipGetDeviceProperties(&prop, dev) );

  // get nvml device by cuda device
  char cuda_pci_busId[NVML_DEVICE_PCI_BUS_ID_BUFFER_SIZE];
  CHECK_CUDA( hipDeviceGetPCIBusId( cuda_pci_busId, NVML_DEVICE_PCI_BUS_ID_BUFFER_SIZE, dev ) );
  CHECK_NVML( nvmlDeviceGetHandleByPciBusId( cuda_pci_busId, &device ) );
  nvmlPciInfo_t pci;
  CHECK_NVML( nvmlDeviceGetPciInfo(device, &pci));
//  nvmlClockType_t clock_type = NVML_CLOCK_GRAPHICS; // NVML_CLOCK_MEM
//  nvmlClockId_t clock_id = NVML_CLOCK_ID_CURRENT; // NVML_CLOCK_ID_APP_CLOCK_TARGET
  unsigned int clock = 0; // in Mhz
  unsigned int clock_current = 0; // in Mhz
  unsigned int clock_target = 0; // in Mhz
  unsigned int clock_mem = 0; // in Mhz
  unsigned int clock_current_mem = 0; // in Mhz
  unsigned int clock_target_mem = 0; // in Mhz
  CHECK_NVML(nvmlDeviceGetClockInfo(device, NVML_CLOCK_MEM, &clock_mem)); // current clock
  CHECK_NVML(nvmlDeviceGetClock(device, NVML_CLOCK_MEM, NVML_CLOCK_ID_CURRENT, &clock_current_mem));
  CHECK_NVML(nvmlDeviceGetClock(device, NVML_CLOCK_MEM, NVML_CLOCK_ID_APP_CLOCK_TARGET, &clock_target_mem));
  CHECK_NVML(nvmlDeviceGetClockInfo(device, NVML_CLOCK_GRAPHICS, &clock)); // current clock
  CHECK_NVML(nvmlDeviceGetClock(device, NVML_CLOCK_GRAPHICS, NVML_CLOCK_ID_CURRENT, &clock_current));
  CHECK_NVML(nvmlDeviceGetClock(device, NVML_CLOCK_GRAPHICS, NVML_CLOCK_ID_APP_CLOCK_TARGET, &clock_target));
  if(clock!=clock_current || clock_mem != clock_current_mem)
    std::cerr << ">> nvml current clock mismatch\n";
  int smi_id;
  std::vector<std::string>::const_iterator it = std::find(bus_ids.begin(), bus_ids.end(), std::string(pci.busId));
  if(it==bus_ids.end())
    smi_id = -1;
  else
    smi_id = it-bus_ids.begin();
  std::cout << dev << ": " << prop.name << "\n"
            << " GraphicsClock CUDA & NVML (target clock):   " << prop.clockRate/1000<< " & " << clock << " MHz" << "( @ " << clock_target << " MHz)"
            << "\n MemClock CUDA & NVML (target clock):        " << prop.memoryClockRate/1000 << " & " << clock_mem  << " MHz" << "( @ " << clock_target_mem << " MHz)"
            << "\n PCI-BUS ID:                                 " << pci.busId
            << "\n nvidia-smi ID (for flag '-i'):              " << smi_id
            << "\n";
}

int main() {
  //cuda_info();
  std::vector<std::string> bus_ids;
  std::cout << "--- ALL NVML/nvidia-smi Devices -----\n";
  nvml_info(bus_ids);
  std::cout << "--- Visible CUDA Devices -----\n";
  int ndevs = 0;
  CHECK_CUDA( hipGetDeviceCount(&ndevs) );

  for(int i=0; i<ndevs; ++i) {
    //test_cuda_and_nvml_device(i);
    CHECK_CUDA(hipSetDevice(i));
    CHECK_NVML( nvmlInit() );
    /**
     * hipFree(0) forces context and clock setting initialization on GPU
     *
     * hipSetDevice or hipMalloc are not enough to apply clock settings.
     * If omitted, first kernel would start with different setting.
     * Also place this after nvmlInit() to get current clock settings.
     */
    CHECK_CUDA( hipFree(0) );
    show_freqs(i, bus_ids);
    CHECK_NVML(nvmlShutdown());
  }

  return 0;
}
